#include "hip/hip_runtime.h"
/*
    CS 4402 Distributed and Parallel Systems
    Assignment 2 Question 1: N thread blocks and N threads per thread block
    Sarah Whelan 250778849
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

int modBy = 103; // common prime num used for modding coefficient values during generation, multiplication, and addition

void genPolynomials(int *polyA, int *polyB, int size);
void multPolynomialsSerial(int *polyA, int *polyB, int polySize, int *product, int productSize);
__global__ void multPolynomialsParallel(int *polyA, int *polyB, int *product, int polySize, int modBy);
__global__ void sumProductsParallel(int prodSize, int threadsPerBlock, int *summedProduct, int *products, int numBlocks, int modBy);
void checkCUDAError(const char* msg);

int main() {
    srand(time(NULL));
    int numTerms;

    // get user desired input on length of polynomials
    printf("Specify the number of terms in the polynomial by specifying the exponent on base 2 UP TO 10, e.g. type 3 if you want 2^3 terms per polynomial: ");
    scanf("%d", &numTerms);

    printf("Value entered is %d\n", numTerms);
    if (numTerms > 10) {
        printf("Invalid entry. The maximum number of terms is 2^10. Please enter a term less than or equal to 10 next time.");
        return 0;
    }
    
    // then bitshift by input value to determine actual value of numTerms
    numTerms = 1 << numTerms;
    printf("Number of terms per polynomial is %d, hence each polynomial has degree %d\n\n", numTerms, numTerms-1);

    // use numTerms as the number of blocks per thread and the number of blocks
    int threadsPerBlock = numTerms;
    int blocks = numTerms;

    // instantiate and allocate host memory blocks to store each polynomial of size numTerms
    int *host_polyA, *host_polyB;
    host_polyA = (int *) malloc(numTerms * sizeof(int));
    host_polyB = (int *) malloc(numTerms * sizeof(int));

    // generate random polynomials of size numTerms
    genPolynomials(host_polyA, host_polyB, numTerms);

    printf("polyA:\n");
    for (int i = 0; i < numTerms; i++) {
        printf("%dx^%d ", host_polyA[i], i);
        if (i != numTerms-1) {
            printf("+ ");
        }
    }

    printf("\n\npolyB:\n");
    for (int i = 0; i < numTerms; i++) {
        printf("%dx^%d ", host_polyB[i], i);
        if (i != numTerms-1) {
            printf("+ ");
        }
    }

    printf("\n\n");

    // determine degree of product
    int degreeOfProduct = (numTerms - 1) * 2; // e.g. degree(polyA, polyB) = 3 then x^3 * x^3 = x^6 and degree = numTerms - 1

    // allocate blocks of memory on the host for storing the product with size degreeOfProduct + 1 (serial)
    // and numTerms*numTerms for the intermediary parallel product, as well asthe final parallel product
    // two different allocations in order to verify results at the end!
    int *host_product_serial, *host_product_parallel, *host_final_product;
    host_product_serial = (int *) malloc((degreeOfProduct+1) * sizeof(int)); // sum of products is intrinsic
    host_product_parallel = (int *) malloc(numTerms * numTerms * sizeof(int)); // because of n threads in each n thread blocks
    host_final_product = (int *) malloc((degreeOfProduct+1) * sizeof(int)); // final product from parallel version once summed

    // ensure all vals in host_product_parallel are 0 (this is done within the serial version so we don't need to worry about that one)
    for (int i = 0; i < numTerms*numTerms; i++) {
        host_product_parallel[i] = 0;
    }
    // ensure all vals in host_final_product are 0
    for (int i = 0; i < degreeOfProduct+1; i++) {
        host_final_product[i] = 0;
    }

    // initialize and allocate memory on the devices for storing dev_polyA, dev_polyB, and dev_product
    int *dev_polyA, *dev_polyB, *dev_product;
    hipMalloc( (void **) &dev_polyA, numTerms * sizeof(int));
    hipMalloc( (void **) &dev_polyB, numTerms * sizeof(int));
    hipMalloc( (void **) &dev_product, numTerms * numTerms * sizeof(int));

    // copy polynomials: host -> device (dest, src, size, direction)
    hipMemcpy(dev_polyA, host_polyA, numTerms * sizeof(int), hipMemcpyHostToDevice); 
    hipMemcpy(dev_polyB, host_polyB, numTerms * sizeof(int), hipMemcpyHostToDevice); 
    hipMemcpy(dev_product, host_product_parallel, numTerms * numTerms * sizeof(int), hipMemcpyHostToDevice);

    // setup kernel params & launch
    dim3 dimGrid(blocks);
    dim3 dimBlock(threadsPerBlock);
    multPolynomialsParallel<<<dimGrid, dimBlock>>>(dev_polyA, dev_polyB, dev_product, numTerms, modBy);

    hipDeviceSynchronize(); // wait for ALL threads from all blocks to complete
    checkCUDAError("kernel invocation");

    // copy dev_product back into host_product_parallel (dest, src, size, direction)
    hipMemcpy(host_product_parallel, dev_product, numTerms * numTerms * sizeof(int), hipMemcpyDeviceToHost);
    
    /* ~~~ now we need to deal with the summation of intermediary products ~~~ */

    // allocate device mem for final product
    int *dev_final;
    hipMalloc( (void **) &dev_final, (degreeOfProduct+1) * sizeof(int));

    // copy zero'd host_final_product to dev_final (dest, src, size, direction) and host_product_parallel to dev_product
    cudaMemcy(dev_final, host_final_product, (degreeOfProduct+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_product, host_product_parallel, numTerms * numTerms * sizeof(int), hipMemcpyHostToDevice);

    // parameters are: int prodSize, int threadsPerBlock, int *summedProduct, int *products, int numBlocks, int modBy)
    sumProductsParallel<<<dimGrid, dimBlock>>>(degreeOfProduct+1, threadsPerBlock, dev_final, dev_product, blocks, modBy);

    hipDeviceSynchronize(); // wait for ALL threads from all blocks to complete
    checkCUDAError("kernel invocation");

    // copy summation of products back to host (dest, src, size, direction)
    hipMemcpy(host_final_product, dev_final, (degreeOfProduct+1) * sizeof(int), hipMemcpyDeviceToHost);

    // multiply polynomials in serial and write to host_product_serial for verification
    multPolynomialsSerial(host_polyA, host_polyB, numTerms, host_product_serial, degreeOfProduct + 1);

    printf("serial result:\n");
    for (int i = 0; i < degreeOfProduct+1; i++) {
        printf("%dx^%d ", host_product_serial[i], i);
        if (i != degreeOfProduct) {
            printf("+ ");
        }
    }
    printf("\n\nparallel result:\n");
    for (int i = 0; i < degreeOfProduct; i++) {
        printf("%dx^%d ", host_final_product[i], i);
        if (i != degreeOfProduct) {
            printf("+ ");
        }
    }
    printf("\n\nequal??? ");
    for (int i = 0; i < degreeOfProduct+1; i++) {
        if (host_product_serial[i] == host_final_product[i]) {
            printf("Y ");
        } else {
            printf("N ");
        }
    }

    // free host and device memory
    free(host_polyA);
    free(host_polyB);
    free(host_product_serial);
    free(host_final_product);

    hipFree(dev_polyA);
    hipFree(dev_polyB);
    hipFree(dev_product);

    return 0;
}

// genPolynomials takes two polynomials and their size (number of terms per polynomial),
// and generates random coefficients for each term mod p
void genPolynomials(int *polyA, int *polyB, int size) {

    // coefficient generation using rand mod p where p = 103
    for (int i = 0; i < size; i++) {
        polyA[i] = rand() % modBy;
        if (polyA[i] == 0) {
            polyA[i] = 1;
        }

        polyB[i] = rand() % modBy;
        if (polyB[i] == 0) {
            polyB[i] = 1;
        }
    }
}

// multPolynomialsSerial takes two polynomials and their size, in addition to a memory block to place 
// the sum of products into, as well as the size of the product polynomial
void multPolynomialsSerial(int *polyA, int *polyB, int polySize, int *product, int productSize) {
    int degreeOfTerms;

    // ensure all coefficients of product are 0
    for (int i = 0; i < productSize; i++) {
        product[i] = 0;
    }

    // calculate sum of products
    for (int a = 0; a < polySize; a++) { // iterate through terms in A
        for (int b = 0; b < polySize; b++) { // for each term in A, iterate through all terms in B
            // add degrees (indices) to determine which index this product belongs to in the product array block
            degreeOfTerms = a + b;

            // add product of terms to previous sum and mod by 103
            product[degreeOfTerms] = (product[degreeOfTerms] + polyA[a] * polyB[b]) % modBy;
        }
    }
}

// multPolynomialsParallel determines the intermediary products of the polynomial multiplication problem
__global__ void multPolynomialsParallel(int *polyA, int *polyB, int *product, int polySize, int modBy) {
    int a = blockIdx.x; // all threads in the same block will access the same polyA element
    int b = threadIdx.x; // but all threads will access individual polyB elements
    int myIndex = blockDim.x * blockIdx.x + threadIdx.x; // where to write this thread's product
    product[myIndex] = (polyA[a] * polyB[b]) % modBy;
}

// sumProductsParallel 
__global__ void sumProductsParallel(int prodSize, int threadsPerBlock, int *summedProduct, int *products, int numBlocks, int modBy) {
    int responsibleFor = blockIdx.x * blockDim.x + threadId.x; // used to check which threads are going to be active during this step

    if (responsibleFor < prodSize) { // e.g. if 1 < 7 then this thread is going to be in charge of summing x^1 terms, else will not be active for the remainder
        for (int blockNum = 0; blockNum < numBlocks; blockNum++) {
            for (int indexInBlock = 0; i < threadsPerBlock; indexInBlock++) {
                int degreeOfElement = blockNum + indexInBlock;
                if (degreeOfElement == responsibleFor) {
                    int spotInProducts = blockNum * blockDim.x + indexInBlock;
                    summedProduct[responsibleFor] = (summedProduct[responsibleFor] + products[spotInProducts]) % modBy;
                }
            }
        }
    }
}

void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if(hipSuccess != err)
    {
        fprintf(stderr, "CUDA error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

