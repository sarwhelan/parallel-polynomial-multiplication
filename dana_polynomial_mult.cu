
#include <hip/hip_runtime.h>
#include <cstdio>

using namespace std;

const int MAX_COEFF = 103;

void random_polynomial(int* p,  int n)
{
    for (int i=0; i<n; i++) {
        p[i] = rand() % MAX_COEFF;
    }
}

__global__ void calculate_products(int *prods, int *x, int *y, int t, size_t n) 
{
    int index = blockIdx.x * blockDim.x;
    for (int i = 0; i < t; i++)
    {
        for (int j = 0; j < t; j++)
        {
            prods[index] += x[i] * y[j];
        }
    }
}

__global__ void reduce_polynomial(int *prods, int *ans, size_t n)
{
    // combine like terms
    int i, j;
    if (blockIdx.x <= (2*n-2)/2)
    {
        i = blockIdx.x, j = 0;
    }
    else
    {
        i = n-1, j = (blockIdx.x % n) + 1;
    }
    while (i >= 0 && j < n)
    {
        ans[blockIdx.x] += prods[i*n + j];
        i--;
        j++;
    }
}

int main() {
    const int n = 1024;
    const int t = 64;    
    int *X = NULL;
    int *Y = NULL;
    int *P = NULL; // products
    int *Poly = NULL;
    X = new int[n];
    Y = new int[n];
    P = new int[n*n];
    Poly = new int[2*n-1];

    random_polynomial(X, n);
    random_polynomial(Y, n);

    for (int i = 0; i < n*n; i++)
    {
        P[i] = 0;
    }
    for (int i = 0; i < 2*n-1; i++)
    {
        Poly[i] = 0;
    }

    // Products
	
	int *Xd, *Yd, *Pd;
	hipMalloc((void **)&Xd, sizeof(int)*n);
    hipMalloc((void **)&Yd, sizeof(int)*n);
    hipMalloc((void **)&Pd, sizeof(int)*n*n);

	hipMemcpy(Xd, X, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(Yd, Y, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(Pd, P, sizeof(int)*n*n, hipMemcpyHostToDevice);

	calculate_products<<<(n*n)/t, t>>>(Pd, Xd, Yd, t, n);
    
    // Sums to final polynomial

    int *Polyd;
    hipMalloc((void **)&Polyd, sizeof(int)*2*n-1);

    hipMemcpy(Polyd, Poly, sizeof(int)*2*n-1, hipMemcpyHostToDevice);

    // START REDUCTION KERNEL HERE AND JUST FOR-LOOP THRU THE BLOCK
    reduce_polynomial<<<2*n-1, 1>>>(Pd, Polyd, n);
    hipMemcpy(Poly, Polyd, sizeof(int)*2*n-1, hipMemcpyDeviceToHost);

	    // Print input, output
    for (int i = 0; i < n; ++i) printf("%2d ", X[i]);
    printf("\n\n");
    for (int i = 0; i < n; ++i) printf("%2d ", Y[i]);
    printf("\n\n");
    for (int i = 0; i < 2*n-1; ++i) printf("%2d ", Poly[i]);
    printf("\n\n");
    
    delete [] X;
    delete [] Y;
    delete [] P;
    delete [] Poly;
	
	hipFree(Xd);
    hipFree(Yd);
    hipFree(Pd);
    hipFree(Polyd);
	
	return 0;
}
